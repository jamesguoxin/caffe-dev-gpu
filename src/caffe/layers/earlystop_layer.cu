#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <limits>
#include <fstream>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe{
    
    template <typename Dtype>
    void EarlystopLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                            const vector<Blob<Dtype>*>& top) {
        Dtype* bottom_data = bottom[0]->mutable_cpu_data();
        Dtype* top_data = top[0]->mutable_cpu_data();
        Dtype* bottom_diff = bottom[0]->mutable_cpu_diff();
        Dtype tmp = 0;
        Dtype val_loss_mean = 0;
        top_data[0] = bottom_data[0];
        if (Caffe::phase() == Caffe::TRAIN) {
            if (index / iter_train_ > time_interval_ && index % iter_train_ == 1) {         // Only start to check after time_interval_ epochs
                LOG(INFO) << "Start CHECKING" << std::endl;
                std::ifstream tmp2(path_tmp_.c_str(), std::ios::in | std::ios::binary);
                tmp2.read(reinterpret_cast<char*>(&val_list[0]), val_list.size()*sizeof(Dtype));
                tmp2.close();
                minimum = EarlystopLayer<Dtype>::find_min(val_list, time_interval_);
                //sum_loss = EarlystopLayer<Dtype>::sum_lastk(val_list, time_interval_);
                LOG(INFO) << "MINIMUM is " << minimum << std::endl;
                for (int idx = 0; idx < val_list.size() - 1; idx++) {
                    if ((val_list[idx] - val_list[idx+1]) > 0) {
                        tmp += fabs(val_list[idx] - val_list[idx+1]) / val_list[idx];
                    }
                }
                tmp = tmp / (val_list.size() - 1);
                tmp = tmp * lamina_;
                LOG(INFO) << "The value for comparison is " << tmp << std::endl;
                if (tmp < threshold_) {
                    stop = true;
                }
                
                if (stop == true) {
                    bottom_diff[0] = 1.5;
                    LOG(INFO) << "Sub-task should be terminated" << std::endl;
                } else {
                    LOG(INFO) << "Sub-task should continue" << std::endl;
                }
            }
            if (index % iter_train_ == 0) {
                train_loss.push_back(bottom_data[0]);
            }
            index = index + 1;
            
        } else if (Caffe::phase() == Caffe::TEST) {
            //LOG(INFO) << "EARLYSTOP Testing phase" << std::endl;
            val_loss.push_back(bottom_data[0]);
            if (val_loss.size() > 0 && val_loss.size() % iter_test_ == 0) {
                val_loss_mean = EarlystopLayer<Dtype>::sum_lastk(val_loss, iter_test_) / iter_test_;
                if (val_list.size() > (val_loss.size() / iter_test_ - 1)) {
                    val_list[val_loss.size() / iter_test_ - 1] = val_loss_mean;
                } else {
                    for (int idx = 0; idx < val_list.size() - 1; idx ++) {
                        val_list[idx] = val_list[idx + 1];
                    }
                    val_list[val_list.size() - 1] = val_loss_mean;
                }
                //LOG(INFO) << "Last value of Val_List is " << val_list[val_list.size() - 1] << std::endl;
                //LOG(INFO) << "Val Loss is " << val_loss_mean << std::endl;
                std::ofstream tmp1(path_tmp_.c_str(), std::ios::out | std::ios::binary);
                tmp1.write(reinterpret_cast<char*>(&val_list[0]), val_list.size()*sizeof(Dtype));
                tmp1.close();
            }
        }
    }
    
    template <typename Dtype>
    void EarlystopLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                             const vector<bool>& propagate_down,
                                             const vector<Blob<Dtype>*>& bottom) {
        LOG(INFO) << "Enter Backward propagation of Earlystop Layer" << std::endl;
        Dtype* bottom_data = bottom[0]->mutable_cpu_diff();
        if (stop == true) {
            bottom_data[0] = 0;
            LOG(INFO) << "EarlyStop Gradient is 0, update will be terminated" << std::endl;
        } else {
            bottom_data[0] = 1;
            LOG(INFO) << "EarlyStop Gradient is 1, update will continue" << std::endl;
        }
    }
    
    INSTANTIATE_LAYER_GPU_FUNCS(EarlystopLayer);
} // namespace caffe
